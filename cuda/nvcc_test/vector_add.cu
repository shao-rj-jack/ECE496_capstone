#define N 10

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

int main(){
    float *a, *b, *out;
    float *d_a, *d_b, *d_out;

    a = (float*)malloc(sizeof(float) * N);
    b = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Allocate device memory for cuda
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);
	
	// Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }
	
    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    
    vector_add<<<1,1>>>(d_out, d_a, d_b, N);
    
	//copy output
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
	
	for(int i = 0; i < N; i++)
		printf("%f," , out[i]);
	printf("\n");
	
    // Cleanup after kernel execution
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    free(a);
    free(b);
    free(out);
	
	return 0;
}
