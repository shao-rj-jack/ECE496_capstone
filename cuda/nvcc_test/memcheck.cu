
#include <hip/hip_runtime.h>
#include<stdint.h>
#include<iostream>

int main(){
    int64_t nvtx_scale = ((int64_t)1)<<14;

    uint64_t* cost = (uint64_t*)malloc(sizeof(uint64_t)*nvtx_scale);

    for(int64_t i=0; i < nvtx_scale; i++)
        cost[i] = uint64_t(123456789);

    uint64_t* dcost;
	hipError_t err;

    err = hipMalloc(&dcost, nvtx_scale*sizeof(uint64_t));
    if(err!= hipSuccess) std::cout << "ERROR " << err << std::endl;

	err = hipMemcpy(dcost, cost, sizeof(uint64_t)*nvtx_scale, hipMemcpyHostToDevice);
	if(err!= hipSuccess) std::cout << "ERROR " << err << std::endl;


    memset(cost, 0, sizeof(uint64_t)*nvtx_scale);
    	
	err = hipMemcpy(cost, dcost, sizeof(uint64_t)*nvtx_scale, hipMemcpyDeviceToHost);
	
	if(err!= hipSuccess) std::cout << "ERROR " << err << std::endl;
    for(int i=0; i<10; i++) {
        std::cout << i << " " << cost[i] << std::endl;
    }

    return 0;
}
